#include "hip/hip_runtime.h"
/**
 * gemm.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <hip/hip_runtime.h>

#include "../../common/polybenchUtilFuncts.h"

#define GPU_DEVICE 0

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

/* Problem size */
#define NI 512
#define NJ 512
#define NK 512

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 16 
#define DIM_THREAD_BLOCK_Y 16 

/* Declared constant values for ALPHA and BETA (same as values in PolyBench 2.0) */
#define ALPHA 32412.0f
#define BETA 2123.0f

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;



void gemm(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C)
{
	int i,j,k;
	
	for (i = 0; i < NI; i++)
	{
    	for (j = 0; j < NJ; j++)
    	{
			C[i*NJ + j] *= BETA;
	
			for (k = 0; k < NK; ++k)
			{
	  			C[i*NJ + j] += ALPHA * A[i*NK + k] * B[k*NJ + j];
			}
      	}
	}
}


void init(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C)
{
	int i, j;

  	for (i = 0; i < NI; i++)
	{
    	for (j = 0; j < NK; j++)
		{
      		A[i*NK + j] = ((DATA_TYPE) i*j) / NI;
		}
	}

  	for (i = 0; i < NK; i++)
	{
    	for (j = 0; j < NJ; j++)
		{
      		B[i*NJ + j] = ((DATA_TYPE) i*j + 1) / NJ;
		}
	}

  	for (i = 0; i < NI; i++)
	{
    	for (j = 0; j < NJ; j++)
		{
      		C[i*NJ + j] = ((DATA_TYPE) i*j + 2) / NJ;
		}
	}
}


void compareResults(DATA_TYPE* C, DATA_TYPE* C_outputFromGpu)
{
	int i, j, fail;
	fail = 0;
	
	// Compare C1 and C2
	for (i=0; i < NI; i++) 
	{
		for (j=0; j < NJ; j++) 
		{
			if (percentDiff(C[i*NJ + j], C_outputFromGpu[i*NJ + j]) > PERCENT_DIFF_ERROR_THRESHOLD) 
			{
				fail++;
			}
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}


__global__ void gemm_kernel(DATA_TYPE *a, DATA_TYPE *b, DATA_TYPE *c)
{
	__shared__ DATA_TYPE s_a[256]; // 16 x 16 
	__shared__ DATA_TYPE s_b[256];

	int bx = blockIdx.x; 
	int by = blockIdx.y;

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	// A : NI x NK
	// B : NK x NJ
	int aBegin = NK * 16 * by;
	int aEnd = aBegin + NK - 1; 
	int aStep = 16;

	int bBegin = bx * 16;
	int bStep = 16 * NJ;

	float sum = 0.f;

	int aa, bb;
	for(aa = aBegin, bb = bBegin; aa <= aEnd; aa += aStep, bb += bStep)
	{
		// load titled sub-matrices into local memory	
		s_a[ty * 16 + tx] = a[aa + ty * NK + tx];
		s_b[ty * 16 + tx] = b[bb + ty * NJ + tx];
		
		__syncthreads();

		int k;
#pragma unroll
		for(k = 0; k < 16; ++k)
		{
			sum += s_a[ty * 16 + k] * s_b[k * 16 + tx];
		}

		__syncthreads();
	}

	c[row * NJ + col] = ALPHA * sum + c[row * NJ + col] * BETA;
}


void gemmCuda(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* C_outputFromGpu)
{
	//double t_start, t_end;

	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;
	DATA_TYPE *C_gpu;

	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NK);
	hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NK * NJ);
	hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * NI * NJ);
	
	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NK, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice);
	hipMemcpy(C_gpu, C, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyHostToDevice);
	
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((size_t)(ceil( ((float)NJ)/ ((float)block.x) )),(size_t)(ceil( ((float)NI)/ ((float)block.y) )));

    hipError_t error;

	hipEvent_t start, stop;

	error = hipEventCreate(&start);
	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to create start event (error code %s)!\n", 
		hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	error = hipEventCreate(&stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to create stop event (error code %s)!\n", 
		hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Record the start event
	error = hipEventRecord(start, NULL);
	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to record start event (error code %s)!\n", 
		hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}



	gemm_kernel<<< grid, block >>>(A_gpu, B_gpu, C_gpu);

	// Record the stop event
	error = hipEventRecord(stop, NULL);
	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to record stop event (error code %s)!\n", 
		hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Wait for the stop event to complete
	error = hipEventSynchronize(stop);
	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", 
		hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	float sgemm_msec = 0.f;
	error = hipEventElapsedTime(&sgemm_msec, start, stop);
	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", 
		hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}



	// C := alpha*op( A )*op( B ) + beta*C
	// GEMM performs 5 floating point operations for one data output
	double flops_sgemm = 5.f * (double) NI * (double) NJ * (double) NK;

	double gigaFlops = (flops_sgemm * 1.0e-9f) / (sgemm_msec / 1000.f);

	printf("Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops, WorkgroupSize= %u threads/block\n",
			gigaFlops,
			sgemm_msec,
			flops_sgemm,
			block.x * block.y);


	hipMemcpy(C_outputFromGpu, C_gpu, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyDeviceToHost);    
	
	hipFree(A_gpu);
	hipFree(B_gpu);
	hipFree(C_gpu);
}
	

int main(int argc, char *argv[])
{
	double t_start, t_end;

	DATA_TYPE* A;
	DATA_TYPE* B;  
	DATA_TYPE* C;  
	DATA_TYPE* C_outputFromGpu; 

	A = (DATA_TYPE*)malloc(NI*NK*sizeof(DATA_TYPE)); 
	B = (DATA_TYPE*)malloc(NK*NJ*sizeof(DATA_TYPE));   
	C = (DATA_TYPE*)malloc(NI*NJ*sizeof(DATA_TYPE)); 
	C_outputFromGpu = (DATA_TYPE*)malloc(NI*NJ*sizeof(DATA_TYPE)); 

	init(A, B, C);
	
	GPU_argv_init();
	
	gemmCuda(A, B, C, C_outputFromGpu);

	t_start = rtclock();	
	gemm(A, B, C);
	t_end = rtclock();
	fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);
	
	compareResults(C, C_outputFromGpu);

	free(A);
	free(B);  
	free(C);  
	free(C_outputFromGpu); 

    	return 0;
}

